
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <random>

#define EPSILON 1e-8f
#define G 6.67e-11f
#define BLOCKSZ 768

using namespace std::chrono;
using timer = high_resolution_clock;

int iters = 2;   // number of iterations for the simulation to run

/**
 * @brief simulated system 
 * includes an array of the position vectors of every body
 * as well as an array of the velocity vectors of every body
 * in 3D space.
 *
 * using a structure of arrays for better performance
 */
typedef struct System {
    float4* p;
    float4* v;
} System;

/**
 * @brief create the simulation by initialize the bodies
 * 
 * @param bods a pointer to body system
 * @param fields the number of total fields we need to fill up
 */
 void init_bodies(float* bods, int fields){
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(100,200);
    std::uniform_int_distribution<int> mass_distribution(3000,9000);
    int on_mass = 0;
    for (int i = 0; i < fields; i++){
        bods[i] = static_cast<float>(distribution(generator));
        if (on_mass == 3){
            bods[i] = static_cast<float>(mass_distribution(generator));
            on_mass = 0;
            continue;
        }
        on_mass++;
    }
}

__global__ void simulate_interaction(float4* p, float4* v, float dt, int n){
    float4 center_obj = { 0.0f, 0.0f, 0.0f, 5000.0f };

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float fx = 0.0f; float fy = 0.0f; float fz = 0.0f;

        for (int j = 0; j < n; j++) {
            
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
            float invDist = rsqrtf(distSqr);
            float denom_cubed = invDist * invDist * invDist;
    
            float m_j = p[j].w;

            fx += m_j * dx * denom_cubed; 
            fy += m_j * dy * denom_cubed; 
            fz += m_j * dz * denom_cubed;
        }

        // calculate interaction with center mass
        float dx = p[i].x - center_obj.x;
        float dy = p[i].y - center_obj.y;
        float dz = p[i].z - center_obj.z;
        float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
        float denom = rsqrtf(d);
        float denom_cubed = denom * denom * denom;

        float m_c = center_obj.w;

        fx -= m_c * dx * denom_cubed; 
        fy -= m_c * dy * denom_cubed; 
        fz -= m_c * dz * denom_cubed;
  
        v[i].x += dt * G * fx;
        v[i].y += dt * G * fy;
        v[i].z += dt * G * fz;
    }
}

#ifdef CHECK
inline void host_interaction(float4* p, float4* v, float dt, int n){
    float4 center_obj = { 0.0f, 0.0f, 0.0f, 5000.0f };

    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < n; i++){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;

        for (int j = 0; j < n; j++){
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
            float denom = rsqrtf(d);
            float denom_cubed = denom * denom * denom;

            float m_j = p[j].w;

            fx += m_j * dx * denom_cubed; 
            fy += m_j * dy * denom_cubed; 
            fz += m_j * dz * denom_cubed;
        }

        // calculate interaction with center mass
        float dx = p[i].x - center_obj.x;
        float dy = p[i].y - center_obj.y;
        float dz = p[i].z - center_obj.z;
        float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
        float denom = rsqrtf(d);
        float denom_cubed = denom * denom * denom;

        float m_c = center_obj.w;

        fx -= m_c * dx * denom_cubed; 
        fy -= m_c * dy * denom_cubed; 
        fz -= m_c * dz * denom_cubed;

        v[i].x += dt * G * fx;
        v[i].y += dt * G * fy;
        v[i].z += dt * G * fz;
    }
}
#endif

int main(int argc, char* argv[]){

    int n = 30000;
    if (argc > 1)
        n = atoi(argv[1]);
    if (argc > 2)
        iters = atoi(argv[2]);
        
    const float dt = 0.01f; // time delta
        
    int bytes = n*2*sizeof(float4);
    float* tmp = (float*) malloc(bytes);
    System bodies = { (float4*) tmp, ((float4*) tmp) + n};  // n is the offset to get to the velocity

    init_bodies(tmp, 8*n);

    // copy bodies for the cpu to use 
    #ifdef CHECK

    float* h_tmp = (float*) malloc(bytes);
    memcpy(h_tmp, tmp, bytes);
    System h_bodies = { (float4*) h_tmp, ((float4*) h_tmp) + n};

    for (int iter = 0; iter < iters; iter++){

        host_interaction(h_bodies.p, h_bodies.v, dt, n);

        // note: OpenMP SIMD is only noticable when compiled with -O1 or -O2
        // because -O3 tries to auto-vectorize loops like these
        #pragma omp simd
        for (int i = 0; i < n; i++){
            h_bodies.p[i].x += h_bodies.v[i].x*dt;
            h_bodies.p[i].y += h_bodies.v[i].y*dt;
            h_bodies.p[i].z += h_bodies.v[i].z*dt;
        }
    }        

    #endif

    float* d_tmp;
    hipMalloc(&d_tmp, bytes);
    System d_bodies = { (float4*) d_tmp, ((float4*) d_tmp) + n};

    int dimGrid = (n + BLOCKSZ - 1)/BLOCKSZ;

    for (int i = 0; i < iters; i++){
        // first kernel launch takes forever
        // https://stackoverflow.com/questions/57709333/cuda-kernel-runs-faster-the-second-time-it-is-run-why

        hipMemcpy(d_tmp, tmp, bytes, hipMemcpyHostToDevice);
        // call kernel
        // #ifndef CHECK
        // cudaEventRecord was giving me zeros all the time. no idea why
        // decided to go with chrono because who cares
        auto start = timer::now();
        // #endif

        simulate_interaction<<<dimGrid, BLOCKSZ>>>(d_bodies.p, d_bodies.v, dt, n);
        hipMemcpy(tmp, d_tmp, bytes, hipMemcpyDeviceToHost);

        #pragma omp simd
        for (int b = 0; b < n; b++){
            bodies.p[b].x += bodies.v[b].x*dt;
            bodies.p[b].y += bodies.v[b].y*dt;
            bodies.p[b].z += bodies.v[b].z*dt;
        }

        // #ifndef CHECK
        auto end = timer::now();
        auto elapsed = duration_cast<microseconds>(end - start).count();
        float elapsed_ms = static_cast<float>(elapsed) / 1000;

        printf("Iter %d took %.2f milliseconds on the device\n", i, elapsed_ms);
        // #endif
    }
    
    #ifdef CHECK
    const float epsilon = 0.0001;
    for (int i = 0; i < n; i++){

        // if (i == 10){
        //     printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
        //     printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
        //     printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
        // }

        if (((abs(bodies.p[i].x) - abs(h_bodies.p[i].x)) > epsilon) ||
            ((abs(bodies.p[i].y) - abs(h_bodies.p[i].y)) > epsilon) ||
            ((abs(bodies.p[i].z) - abs(h_bodies.p[i].z)) > epsilon)){
                printf("Host bodies and GPU bodies mismatch!\n");
                printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
                printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
                printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
            }
    }
    free(h_tmp);
    #endif

    free(tmp);
    hipFree(d_tmp);
}