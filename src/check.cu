
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <random>

#define EPSILON 1e-8f
#define BLOCKSZ 512
#define G 6.67e-11f

using namespace std::chrono;
using timer = high_resolution_clock;

int iters = 2;   // number of iterations for the simulation to run

/**
 * @brief simulated system 
 * includes an array of the position vectors of every body
 * as well as an array of the velocity vectors of every body
 * in 3D space.
 *
 * using a structure of arrays for better performance
 */
typedef struct System {
    float4* p;
    float4* v;
} System;

/**
 * @brief create the simulation by initialize the bodies
 * 
 * @param bods a pointer to body system
 * @param fields the number of total fields we need to fill up
 */
 void init_bodies(float* bods, int fields){
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(100,200);
    std::uniform_int_distribution<int> mass_distribution(3000,9000);
    int on_mass = 0;
    for (int i = 0; i < fields; i++){
        if (on_mass == 3){
            bods[i] = static_cast<float>(mass_distribution(generator));
            on_mass = 0;
            continue;
        }
        else {
            bods[i] = static_cast<float>(distribution(generator));
        }
        on_mass++;
    }
}

__global__ void simulate_interaction(float4* p, float4* v, float dt, int n){
    float4 center_obj = { 0.0f, 0.0f, 0.0f, 5000.0f };
    int b = blockDim.x * blockIdx.x + threadIdx.x;
    if (b < n){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;
        
        // iterate over all the other bodies in the simulation
        // this means iterating over the whole grid
        for (int t = 0; t < gridDim.x; t++){
            __shared__ float4 others[BLOCKSZ];
            float4 curr = p[t * blockDim.x + threadIdx.x];
            // load other threads' info into shared memory
            others[threadIdx.x] = make_float4(curr.x, curr.y, curr.z, curr.w);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCKSZ; j++){
                float dx = others[j].x - p[b].x;
                float dy = others[j].y - p[b].y;
                float dz = others[j].z - p[b].z;
                float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
                float denom = rsqrtf(d);
                float denom_cubed = denom * denom * denom;

                float m_j = others[j].w;

                fx += m_j * dx * denom_cubed; 
                fy += m_j * dy * denom_cubed; 
                fz += m_j * dz * denom_cubed;
            }
            __syncthreads();
        }       
        
        // calculate interaction with center mass
        float dx = p[b].x - center_obj.x;
        float dy = p[b].y - center_obj.y;
        float dz = p[b].z - center_obj.z;
        float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
        float denom = rsqrtf(d);
        float denom_cubed = denom * denom * denom;

        float m_c = center_obj.w;

        fx -= m_c * dx * denom_cubed; 
        fy -= m_c * dy * denom_cubed; 
        fz -= m_c * dz * denom_cubed;

        v[b].x += dt * G * fx;
        v[b].y += dt * G * fy;
        v[b].z += dt * G * fz;

        p[b].x += v[b].x*dt;
        p[b].y += v[b].y*dt;
        p[b].z += v[b].z*dt;
    }
}

#ifdef CHECK
inline void host_interaction(float4* p, float4* v, float dt, int n){
    float4 center_obj = { 0.0f, 0.0f, 0.0f, 5000.0f };

    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < n; i++){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;

        for (int j = 0; j < n; j++){
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
            float denom = rsqrtf(d);
            float denom_cubed = denom * denom * denom;

            float m_j = p[j].w;

            fx += m_j * dx * denom_cubed; 
            fy += m_j * dy * denom_cubed; 
            fz += m_j * dz * denom_cubed;
        }

        // calculate interaction with center mass
        float dx = p[i].x - center_obj.x;
        float dy = p[i].y - center_obj.y;
        float dz = p[i].z - center_obj.z;
        float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
        float denom = rsqrtf(d);
        float denom_cubed = denom * denom * denom;

        float m_c = center_obj.w;

        fx -= m_c * dx * denom_cubed; 
        fy -= m_c * dy * denom_cubed; 
        fz -= m_c * dz * denom_cubed;

        v[i].x += dt * G * fx;
        v[i].y += dt * G * fy;
        v[i].z += dt * G * fz;

        p[i].x += v[i].x*dt;
        p[i].y += v[i].y*dt;
        p[i].z += v[i].z*dt;
    }
}
#endif

int main(int argc, char* argv[]){

    int n = 30000;
    if (argc > 1)
        n = atoi(argv[1]);
    if (argc > 2)
        iters = atoi(argv[2]);
        
    const float dt = 0.01f; // time delta
        
    int bytes = n*2*sizeof(float4);
    float* tmp = (float*) malloc(bytes);
    System bodies = { (float4*) tmp, ((float4*) tmp) + n};  // n is the offset to get to the velocity

    init_bodies(tmp, 8*n);

    // copy bodies for the cpu to use 
    #ifdef CHECK

    float* h_tmp = (float*) malloc(bytes);
    memcpy(h_tmp, tmp, bytes);
    System h_bodies = { (float4*) h_tmp, ((float4*) h_tmp) + n};

    for (int iter = 0; iter < iters; iter++){

        host_interaction(h_bodies.p, h_bodies.v, dt, n);

        // note: OpenMP SIMD is only noticable when compiled with -O1 or -O2
        // because -O3 tries to auto-vectorize loops like these
        // #pragma omp simd
        // for (int i = 0; i < n; i++){
        //     h_bodies.p[i].x += h_bodies.v[i].x*dt;
        //     h_bodies.p[i].y += h_bodies.v[i].y*dt;
        //     h_bodies.p[i].z += h_bodies.v[i].z*dt;
        // }
    }        

    #endif

    float* d_tmp;
    hipMalloc(&d_tmp, bytes);
    System d_bodies = { (float4*) d_tmp, ((float4*) d_tmp) + n};

    int dimGrid = (n + BLOCKSZ - 1)/BLOCKSZ;

    for (int i = 0; i < iters; i++){
        // first kernel launch takes forever
        // https://stackoverflow.com/questions/57709333/cuda-kernel-runs-faster-the-second-time-it-is-run-why

        hipMemcpy(d_tmp, tmp, bytes, hipMemcpyHostToDevice);
        // call kernel
        #ifndef CHECK
        // cudaEventRecord was giving me zeros all the time. no idea why
        // decided to go with chrono because who cares
        auto start = timer::now();
        #endif

        simulate_interaction<<<dimGrid, BLOCKSZ>>>(d_bodies.p, d_bodies.v, dt, n);
        hipMemcpy(tmp, d_tmp, bytes, hipMemcpyDeviceToHost);

        // #pragma omp simd
        // for (int b = 0; b < n; b++){
        //     bodies.p[b].x += bodies.v[b].x*dt;
        //     bodies.p[b].y += bodies.v[b].y*dt;
        //     bodies.p[b].z += bodies.v[b].z*dt;
        // }

        #ifndef CHECK
        auto end = timer::now();
        auto elapsed = duration_cast<microseconds>(end - start).count();
        float elapsed_ms = static_cast<float>(elapsed) / 1000;

        printf("Iter %d took %.2f milliseconds on the device\n", i, elapsed_ms);
        #endif
    }

    #ifdef CHECK
    const float epsilon = 0.0001;
    for (int i = 0; i < n; i++){

        if (i == 10){
            printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
            printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
            printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
        }
        // if (h_bodies.p[i].x > 200 || 
        //     h_bodies.p[i].y > 200 || 
        //     h_bodies.p[i].z > 200){
        //         printf("stuff is working haha yes\n");
        //     }

        if ((abs(abs(bodies.p[i].x) - abs(h_bodies.p[i].x)) > epsilon) ||
            (abs(abs(bodies.p[i].y) - abs(h_bodies.p[i].y)) > epsilon) ||
            (abs(abs(bodies.p[i].z) - abs(h_bodies.p[i].z)) > epsilon)){
                printf("Host bodies and GPU bodies mismatch!\n");
                printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
                printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
                printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
            }
    }
    free(h_tmp);
    #endif

    free(tmp);
    hipFree(d_tmp);
}