
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <random>

#define EPSILON 1e-8f
#define BLOCKSZ 1024
#define G 6.67e-11f

using namespace std::chrono;
using timer = high_resolution_clock;

int iters = 2;   // number of iterations for the simulation to run

/**
 * @brief simulated system 
 * includes an array of the position vectors of every body
 * as well as an array of the velocity vectors of every body
 * in 3D space.
 *
 * using a structure of arrays for better performance
 */
typedef struct System {
    float4* p;
    float4* v;
} System;

/**
 * @brief create the simulation by initialize the bodies
 * 
 * @param bods a pointer to body system
 * @param fields the number of total fields we need to fill up
 */
 void init_bodies(float* bods, int fields){
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(0,9);
    for (int i = 0; i < fields; i++){
        bods[i] = static_cast<float>(distribution(generator));
    }
}

__global__ void simulate_interaction(float4* p, float4* v, float dt, int n){
    int b = blockDim.x * blockIdx.x + threadIdx.x;
    if (b < n){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;
        
        // iterate over all the other bodies in the simulation
        // this means iterating over the whole grid
        for (int t = 0; t < gridDim.x; t++){
            __shared__ float4 others[BLOCKSZ];
            float4 curr = p[t * blockDim.x + threadIdx.x];
            // load other threads' info into shared memory
            others[threadIdx.x] = make_float4(curr.x, curr.y, curr.z, curr.w);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCKSZ; j++){
                float dx = others[j].x - p[b].x;
                float dy = others[j].y - p[b].y;
                float dz = others[j].z - p[b].z;
                float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
                float denom = rsqrtf(d);
                float denom_cubed = denom * denom * denom;

                float m_j = others[j].w;

                fx += m_j * dx * denom_cubed; 
                fy += m_j * dy * denom_cubed; 
                fz += m_j * dz * denom_cubed;
            }
            __syncthreads();
        }
        v[b].x += dt * G * fx;
        v[b].y += dt * G * fy;
        v[b].z += dt * G * fz;
    }
}

#ifdef CHECK
inline void host_interaction(float4* p, float4* v, float dt, int n){
    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < n; i++){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;

        for (int j = 0; j < n; j++){
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float d = dx*dx + dy*dy + dz*dz + EPSILON * EPSILON;
            float denom = rsqrtf(d);
            float denom_cubed = denom * denom * denom;

            float m_j = p[j].w;

            fx += m_j * dx * denom_cubed; 
            fy += m_j * dy * denom_cubed; 
            fz += m_j * dz * denom_cubed;
        }
        v[i].x += dt * G * fx;
        v[i].y += dt * G * fy;
        v[i].z += dt * G * fz;
    }
}
#endif

int main(int argc, char* argv[]){

    int n = 30000;
    if (argc > 1)
        n = atoi(argv[1]);
    if (argc > 2)
        iters = atoi(argv[2]);
        
    const float dt = 0.01f; // time delta
        
    int bytes = n*2*sizeof(float4);
    float* tmp = (float*) malloc(bytes);
    System bodies = { (float4*) tmp, ((float4*) tmp) + n};  // n is the offset to get to the velocity

    init_bodies(tmp, 8*n);

    // copy bodies for the cpu to use 
    #ifdef CHECK

    float* h_tmp = (float*) malloc(bytes);
    memcpy(h_tmp, tmp, bytes);
    System h_bodies = { (float4*) h_tmp, ((float4*) h_tmp) + n};

    for (int iter = 0; iter < iters; iter++){

        host_interaction(h_bodies.p, h_bodies.v, dt, n);

        // note: OpenMP SIMD is only noticable when compiled with -O1 or -O2
        // because -O3 tries to auto-vectorize loops like these
        #pragma omp simd
        for (int i = 0; i < n; i++){
            h_bodies.p[i].x += h_bodies.v[i].x*dt;
            h_bodies.p[i].y += h_bodies.v[i].y*dt;
            h_bodies.p[i].z += h_bodies.v[i].z*dt;
        }
    }        

    #endif

    float* d_tmp;
    hipMalloc(&d_tmp, bytes);
    System d_bodies = { (float4*) d_tmp, ((float4*) d_tmp) + n};

    int dimGrid = (n + BLOCKSZ - 1)/BLOCKSZ;

    for (int i = 0; i < iters; i++){
        // first kernel launch takes forever
        // https://stackoverflow.com/questions/57709333/cuda-kernel-runs-faster-the-second-time-it-is-run-why

        hipMemcpy(d_tmp, tmp, bytes, hipMemcpyHostToDevice);
        // call kernel
        #ifndef CHECK
        // cudaEventRecord was giving me zeros all the time. no idea why
        // decided to go with chrono because who cares
        auto start = timer::now();
        #endif

        simulate_interaction<<<dimGrid, BLOCKSZ>>>(d_bodies.p, d_bodies.v, dt, n);
        hipMemcpy(tmp, d_tmp, bytes, hipMemcpyDeviceToHost);

        #pragma omp simd
        for (int b = 0; b < n; b++){
            bodies.p[b].x += bodies.v[b].x*dt;
            bodies.p[b].y += bodies.v[b].y*dt;
            bodies.p[b].z += bodies.v[b].z*dt;
        }

        #ifndef CHECK
        auto end = timer::now();
        auto elapsed = duration_cast<microseconds>(end - start).count();
        float elapsed_ms = static_cast<float>(elapsed) / 1000;

        printf("Iter %d took %.2f milliseconds on the device\n", i, elapsed_ms);
        #endif
    }

    #ifdef CHECK
    const float epsilon = 0.0001;
    for (int i = 0; i < n; i++){
        if (((abs(bodies.p[i].x) - abs(h_bodies.p[i].x)) > epsilon) ||
            ((abs(bodies.p[i].y) - abs(h_bodies.p[i].y)) > epsilon) ||
            ((abs(bodies.p[i].z) - abs(h_bodies.p[i].z)) > epsilon)){
                printf("Host bodies and GPU bodies mismatch!\n");
                printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
                printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
                printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
            }
    }
    free(h_tmp);
    #endif

    free(tmp);
    hipFree(d_tmp);
}