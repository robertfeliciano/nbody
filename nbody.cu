
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>

#define EPSILON 1e-8f
#define BLOCKSZ 1024

using namespace std::chrono;
using timer = high_resolution_clock;

const int iters = 10;   // number of iterations for the simulation to run

/**
 * @brief simulated system 
 * includes an array of the position vectors of every body
 * as well as an array of the velocity vectors of every body
 * in 3D space.
 *
 * using a structure of arrays for better performance
 */
typedef struct System {
    float4* p;
    float4* v;
} System;

/**
 * @brief create the simulation by initialize the bodies
 * 
 * @param bods a pointer to body system
 * @param fields the number of total fields we need to fill up
 */
void init_bodies(float* bods, int fields){
    for (int i = 0; i < fields; i++){
        bods[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void simulate_interaction(float4* p, float4* v, float dt, int n){
    int b = blockDim.x * blockIdx.x + threadIdx.x;
    if (b < n){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;
        
        // iterate over all the other bodies in the simulation
        // this means iterating over the whole grid
        for (int t = 0; t < gridDim.x; t++){
            __shared__ float3 others[BLOCKSZ];
            float4 curr = p[t * blockDim.x + threadIdx.x];
            // load other threads' info into shared memory
            others[threadIdx.x] = make_float3(curr.x, curr.y, curr.z);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCKSZ; j++){
                float dx = others[j].x - p[b].x;
                float dy = others[j].y - p[b].y;
                float dz = others[j].z - p[b].z;
                float d = dx*dx + dy*dy + dz*dz + EPSILON;
                float denom = rsqrtf(d);
                float denom_cubed = denom * denom * denom;

                fx += dx * denom_cubed; 
                fy += dy * denom_cubed; 
                fz += dz * denom_cubed;
            }
            __syncthreads();
        }
        v[b].x += dt*fx;
        v[b].y += dt*fy;
        v[b].z += dt*fz;
    }
}

#ifdef CHECK
inline void host_interaction(float4* p, float4* v, float dt, int n){
    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < n; i++){
        // forces in the x, y, z direction
        float fx = 0.0f, fy = 0.0f, fz = 0.0f;

        #pragma unroll
        for (int j = 0; j < n; j++){
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float d = dx*dx + dy*dy + dz*dz + EPSILON;
            float denom = rsqrtf(d);
            float denom_cubed = denom * denom * denom;

            fx += dx * denom_cubed; 
            fy += dy * denom_cubed; 
            fz += dz * denom_cubed;
        }

        v[i].x += dt*fx;
        v[i].y += dt*fy;
        v[i].z += dt*fz;
    }
}
#endif

int main(int argc, char* argv[]){

    int n = 30000;
    if (argc > 1)
        n = atoi(argv[1]);
        
    const float dt = 0.01f; // time delta
        
    int bytes = n*2*sizeof(float4);
    float* tmp = (float*) malloc(bytes);
    System bodies = { (float4*) tmp, ((float4*) tmp) + n};  // n is the offset to get to the velocity

    init_bodies(tmp, 8*n);

    // copy bodies for the cpu to use 
    #ifdef CHECK

    float* h_tmp = (float*) malloc(bytes);
    memcpy(h_tmp, tmp, bytes);
    System h_bodies = { (float4*) h_tmp, ((float4*) h_tmp) + n};

    for (int iter = 0; iter < iters; iter++){

        host_interaction(h_bodies.p, h_bodies.v, dt, n);

        // note: OpenMP SIMD is only noticable when compiled with -O1 or -O2
        // because -O3 tries to auto-vectorize loops like these
        #pragma omp simd
        for (int i = 0; i < n; i++){
            h_bodies.p[i].x += h_bodies.v[i].x*dt;
            h_bodies.p[i].y += h_bodies.v[i].y*dt;
            h_bodies.p[i].z += h_bodies.v[i].z*dt;
        }
    }        

    #endif

    float* d_tmp;
    hipMalloc(&d_tmp, bytes);
    System d_bodies = { (float4*) tmp, ((float4*) tmp) + n};

    int dimGrid = (n + BLOCKSZ - 1)/BLOCKSZ;

    for (int i = 0; i < iters; i++){
        // cudaEventRecord was giving me zeros all the time. no idea why
        // decided to go with chrono because who cares

        // first kernel launch takes forever
        // https://stackoverflow.com/questions/57709333/cuda-kernel-runs-faster-the-second-time-it-is-run-why

        hipMemcpy(d_tmp, tmp, bytes, hipMemcpyHostToDevice);
        // call kernel
        #ifndef CHECK
        auto start = timer::now();
        #endif

        simulate_interaction<<<dimGrid, BLOCKSZ>>>(d_bodies.p, d_bodies.v, dt, n);
        hipMemcpy(tmp, d_tmp, bytes, hipMemcpyDeviceToHost);

        #pragma omp simd
        for (int b = 0; b < n; b++){
            bodies.p[b].x += bodies.v[b].x*dt;
            bodies.p[b].y += bodies.v[b].y*dt;
            bodies.p[b].z += bodies.v[b].z*dt;
        }

        #ifndef CHECK
        auto end = timer::now();
        auto elapsed = duration_cast<microseconds>(end - start).count();
        float elapsed_ms = static_cast<float>(elapsed) / 1000;

        printf("Iter %d took %.2f milliseconds on the device\n", i, elapsed_ms);
        #endif
    }

    #ifdef CHECK
    for (int i = 0; i < n; i++){
        if ((bodies.p[i].x != h_bodies.p[i].x) ||
            (bodies.p[i].y != h_bodies.p[i].y) ||
            (bodies.p[i].z != h_bodies.p[i].z)){
                printf("Host bodies and GPU bodies mismatch!\n");
                printf("d_body %d.x = %f,\nh_body %d.x = %f\n", i, bodies.p[i].x, i, h_bodies.p[i].x);
                printf("d_body %d.y = %f,\nh_body %d.y = %f\n", i, bodies.p[i].y, i, h_bodies.p[i].y);
                printf("d_body %d.z = %f,\nh_body %d.z = %f\n", i, bodies.p[i].z, i, h_bodies.p[i].z);
            }
    }
    free(h_tmp);
    #endif

    free(tmp);
    hipFree(d_tmp);
}